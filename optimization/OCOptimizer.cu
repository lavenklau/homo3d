#include "hip/hip_runtime.h"
#include "OCOptimizer.h"
#include "hip/hip_runtime.h"
#include "culib/lib.cuh"
#include "AutoDiff/TensorExpression.h"
#include "cmdline.h"


using namespace homo;
using namespace culib;

template<typename T>
__global__ void update_kernel(int ne,
	const T* sens, T g,
	const T* rhoold, T* rhonew,
	T minRho, T stepLimit, T damp) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= ne) return;
	
	T rho = rhoold[tid];

	T B = -sens[tid] / g;
	if (B < 0) B = 0.01f;
	T newrho = powf(B, damp) * rho;
	//if (tid == 0) {
	//	printf("sens = %.4e  g = %.4e  damp = %.4e  rho =%.4e  newrho = %.4e\n",
	//		sens[tid], g, damp, rho, newrho);
	//}

	if (newrho - rho < -stepLimit) newrho = rho - stepLimit;
	if (newrho - rho > stepLimit) newrho = rho + stepLimit;
	if (newrho < minRho) newrho = minRho;
	if (newrho > 1) newrho = 1;
	rhonew[tid] = newrho;
}



void OCOptimizer::update(const float* sens, float* rho, float volratio) {
	float* newrho;
	hipMalloc(&newrho, sizeof(float) * ne);
	float maxSens = abs(parallel_maxabs(sens, ne));
	printf("max sens = %f\n", maxSens);
	float minSens = 0;
	for (int itn = 0; itn < 20; itn++) {
		float gSens = (maxSens + minSens) / 2;
		size_t grid_size, block_size;
		make_kernel_param(&grid_size, &block_size, ne, 256);
		update_kernel << <grid_size, block_size >> > (ne, sens, gSens, rho, newrho,
			minRho, step_limit, damp);
		hipDeviceSynchronize();
		cuda_error_check;
		float curVol = parallel_sum(newrho, ne) / ne;
		printf("[OC] : g = %.4e   vol = %4.2f%% (Goal %4.2f%%)       \r", gSens, curVol * 100, volratio * 100);
		if (curVol < volratio - 0.0001) {
			maxSens = gSens;
		}
		else if (curVol > volratio + 0.0001) {
			minSens = gSens;
		}
		else {
			break;
		}
	}
	printf("\n");
	hipMemcpy(rho, newrho, sizeof(float) * ne, hipMemcpyDeviceToDevice);
	hipFree(newrho);
}

__device__ bool is_bounded(int p[3], int reso[3]) {
	return p[0] >= 0 && p[0] < reso[0] &&
		p[1] >= 0 && p[1] < reso[1] &&
		p[2] >= 0 && p[2] < reso[2];
}

template<typename Kernel>
__global__ void filterSens_kernel(
	int ne, devArray_t<int, 3> reso, size_t pitchT,
	const float* sens, const float* rho, const float* weightSum, float* newsens, Kernel wfunc) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= ne) return;
	int epos[3] = { tid % reso[0], tid / reso[0] % reso[1], tid / (reso[0] * reso[1]) };
	float wsum = 0;
	int ereso[3] = { reso[0],reso[1],reso[2] };
	Kernel ker = wfunc;
	float sum = 0;
	for (int nei = 0; nei < wfunc.size(); nei++) {
		int offset[3];
		ker.neigh(nei, offset);
		float w = ker.weight(offset);
		int neighpos[3] = { epos[0] + offset[0], epos[1] + offset[1], epos[2] + offset[2] };
		if (ker.is_period()) {
			for (int i = 0; i < 3; i++) neighpos[i] = (neighpos[i] + reso[i]) % reso[i];
		}
		if (is_bounded(neighpos, ereso)) {
			int neighid = neighpos[0] + (neighpos[1] + neighpos[2] * ereso[1]) * pitchT;
			//w /= weightSum[neighid];
			sum += sens[neighid] * rho[neighid] * w;
			wsum += w;
		}
	}
	int eid = epos[0] + (epos[1] + epos[2] * ereso[1]) * pitchT;
	sum /= wsum * rho[eid];
	newsens[eid] = sum;
}

template<typename Kernel>
__global__ void weightSum_kernel(int ne, devArray_t<int, 3> reso, size_t pitchT,
	float* weightSum, Kernel wfunc
) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= ne) return;
	int epos[3] = { tid % reso[0], tid / reso[0] % reso[1], tid / (reso[0] * reso[1]) };
	float wsum = 0;
	int ereso[3] = { reso[0],reso[1],reso[2] };
	Kernel ker = wfunc;
	float sum = 0;
	for (int nei = 0; nei < wfunc.size(); nei++) {
		int offset[3];
		ker.neigh(nei, offset);
		float w = ker.weight(offset);
		int neighpos[3] = { epos[0] + offset[0], epos[1] + offset[1], epos[2] + offset[2] };
		if (ker.is_period()) {
			for (int i = 0; i < 3; i++) neighpos[i] = (neighpos[i] + reso[i]) % reso[i];
		}
		if (is_bounded(neighpos, ereso)) {
			int neighid = neighpos[0] + (neighpos[1] + neighpos[2] * ereso[1]) * pitchT;
			wsum += w;
		}
	}
	int eid = epos[0] + (epos[1] + epos[2] * ereso[1]) * pitchT;
	weightSum[eid] = wsum;
}

void OCOptimizer::filterSens(float* sens, const float* rho, size_t pitchT, int reso[3], float radius)
{
	static float* filterWeightSum = nullptr;
	if (!filterWeightSum) {
		hipMalloc(&filterWeightSum, sizeof(float) * reso[1] * reso[2] * pitchT);
		init_array(filterWeightSum, float(0), reso[1] * reso[2] * pitchT);
	}
	float* newsens;
	hipMalloc(&newsens, sizeof(float) * reso[1] * reso[2] * pitchT);
	radial_convker_t<float, Linear> convker(radius, 0, true, FLAGS_periodfilt);
	devArray_t<int, 3> ereso{ reso[0],reso[1],reso[2] };
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, ne, 256);
	filterSens_kernel << <grid_size, block_size >> > (ne, ereso, pitchT, sens, rho, filterWeightSum, newsens, convker);
	hipDeviceSynchronize();
	cuda_error_check;
	hipMemcpy(sens, newsens, sizeof(float) * reso[1] * reso[2] * pitchT, hipMemcpyDeviceToDevice);
	hipFree(newsens);
}

template<typename Kernel>
__global__ void filterSens_Tensor_kernel(
	TensorView<float> sens, TensorView<float> rho, TensorView<float> newsens, Kernel wfunc) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ne = rho.size();
	int reso[3] = { rho.size(0),rho.size(1),rho.size(2) };
	if (tid >= ne) return;
	int epos[3] = { tid % reso[0], tid / reso[0] % reso[1], tid / (reso[0] * reso[1]) };
	float wsum = 0;
	Kernel ker = wfunc;
	float sum = 0;
	for (int nei = 0; nei < wfunc.size(); nei++) {
		int offset[3];
		ker.neigh(nei, offset);
		float w = ker.weight(offset);
		int neighpos[3] = { epos[0] + offset[0], epos[1] + offset[1], epos[2] + offset[2] };
		if (ker.is_period()) {
			for (int i = 0; i < 3; i++) neighpos[i] = (neighpos[i] + reso[i]) % reso[i];
		}
		if (is_bounded(neighpos, reso)) {
			//int neighid = neighpos[0] + (neighpos[1] + neighpos[2] * ereso[1]) * pitchT;
			//w /= weightSum[neighid];
			sum += sens(neighpos[0], neighpos[1], neighpos[2]) * rho(neighpos[0], neighpos[1], neighpos[2]) * w;
			wsum += w;
		} 
	}
	//int eid = epos[0] + (epos[1] + epos[2] * ereso[1]) * pitchT;
	sum /= wsum * rho(epos[0], epos[1], epos[2]);
	newsens(epos[0], epos[1], epos[2]) = sum;
}

void OCOptimizer::filterSens(Tensor<float> sens, Tensor<float> rho, float radius /*= 2*/) {
	Tensor<float> newsens(rho.getDim());
	newsens.reset(0);
	radial_convker_t<float, Linear> convker(radius, 0, true, false);
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, rho.size(), 256);
	filterSens_Tensor_kernel << <grid_size, block_size >> > (sens.view(), rho.view(), newsens.view(), convker);
	hipDeviceSynchronize();
	cuda_error_check;
	sens.copy(newsens);
}

template<typename T>
__global__ void update_Tensor_kernel(TensorView<T> sens, T g,
	TensorView<T> rhoold, TensorView<T> rhonew,
	T minRho, T stepLimit, T damp) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ne = rhoold.size();
	if (tid >= ne) return;

	T rho = rhoold(tid);

	T B = -sens(tid) / g;
	if (B < 0) B = 0.01f;
	T newrho = powf(B, damp) * rho;

	if (newrho - rho < -stepLimit) newrho = rho - stepLimit;
	if (newrho - rho > stepLimit) newrho = rho + stepLimit;
	if (newrho < minRho) newrho = minRho;
	if (newrho > 1) newrho = 1;
	rhonew(tid) = newrho;
}

void OCOptimizer::update(Tensor<float> sens, Tensor<float> rho, float volratio) {
	Tensor<float> newrho(rho.getDim());
	newrho.reset(0);
	float maxSens = abs(sens.maxabs());
	printf("max sens = %f\n", maxSens);
	float minSens = 0;
	for (int itn = 0; itn < 20; itn++) {
		float gSens = (maxSens + minSens) / 2;
		size_t grid_size, block_size;
		make_kernel_param(&grid_size, &block_size, rho.size(), 256);
		update_Tensor_kernel << <grid_size, block_size >> > (sens.view(), gSens, rho.view(), newrho.view(),
			minRho, step_limit, damp);
		hipDeviceSynchronize();
		cuda_error_check;
		//float curVol = parallel_sum(newrho, ne) / ne;
		float curVol = newrho.Sum() / newrho.size();
		printf("[OC] : g = %.4e   vol = %4.2f%% (Goal %4.2f%%)       \r", gSens, curVol * 100, volratio * 100);
		if (curVol < volratio - 0.0001) {
			maxSens = gSens;
		}
		else if (curVol > volratio + 0.0001) {
			minSens = gSens;
		}
		else {
			break;
		}
	}
	printf("\n");
	rho.copy(newrho);
}


