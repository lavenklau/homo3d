#include "hip/hip_runtime.h"
#include "grid.h"
#include "homoCommon.cuh"

using namespace homo;

__device__ static bool inStrictBound(int pi[3], int cover[3]) {
	return pi[0] > -cover[0] && pi[0] < cover[0] &&
		pi[1] > -cover[1] && pi[1] < cover[1] &&
		pi[2] > -cover[2] && pi[2] < cover[2];
}

//template<int BlockSize = 256>
__global__ void gs_relaxation_otf_kernel_opt(
	int gs_set, float* rholist,
	devArray_t<int, 3> gridCellReso,
	VertexFlags* vflags, CellFlags* eflags,
	float w = 1.f
) {
	__shared__ float LM[5];
	__shared__ float RHO[8][32];
	__shared__ float sumKeU[3][4][32];
	__shared__ float sumKs[3][3];

	// load lam and mu 
	if (threadIdx.x < 5) {
		LM[threadIdx.x] = gLM[threadIdx.x];
	}

	int warpId = threadIdx.x / 32;
	int laneId = threadIdx.x % 32;

	int vid = blockIdx.x * 32 + laneId;
	
	bool fiction = false;

	// to global vertex id
	vid = gs_set == 0 ? vid : gGsVertexEnd[gs_set - 1] + vid;

	fiction = vid >= gGsVertexEnd[gs_set];

	GridVertexIndex indexer(gridCellReso[0], gridCellReso[1], gridCellReso[2]);
	VertexFlags vflag;
	if (!fiction) {
		vflag = vflags[vid];
		fiction = fiction || vflag.is_fiction() || vflag.is_period_padding();
		indexer.locate(vid, vflag.get_gscolor(), gGsVertexEnd);
	}

	// load density field
	for (int i = 0; i < 8; i++) { RHO[i][laneId] = 0; }
	if (!fiction) {
		CellFlags eflag;
#pragma unroll
		for (int i = 0; i < 8; i++) {
			int elementId = indexer.neighElement(i, gGsCellEnd, gGsCellReso).getId();
			if (elementId != -1) {
				eflag = eflags[elementId];
			/*	if (!eflag.is_fiction())*/ RHO[i][laneId] = powf(rholist[elementId], exp_penal[0]);
			} 	
		}
	} 
	__syncthreads();

	int ev[27];
	if (!fiction) {
		for (int i = 0; i < 27; i++) {
			VertexFlags nvflag;
			int vneighId = indexer.neighVertex(i, gGsVertexEnd, gGsVertexReso).getId();
			ev[i] = vneighId;
			if (vneighId != -1) {
				nvflag = vflags[vneighId];
			}
		}
	}

	float KeU[3] = { 0. };
	float u[3];
	if (!fiction) {
		if (warpId == 0) {
			u[0] = gU[0][ev[0]]; u[1] = gU[1][ev[0]]; u[2] = gU[2][ev[0]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[0][laneId]));
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[0][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[0][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[0][laneId]));
			u[0] = gU[0][ev[1]]; u[1] = gU[1][ev[1]]; u[2] = gU[2][ev[1]];
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[0][laneId] + 2.f * RHO[1][laneId]));
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId]));
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId]));
			KeU[2] += u[1] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[1][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[1][laneId]));
			KeU[1] += u[2] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[1][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[1][laneId]));
			u[0] = gU[0][ev[2]]; u[1] = gU[1][ev[2]]; u[2] = gU[2][ev[2]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[1][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[1][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[1][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[1][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[1][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[1][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[1][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[1][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[1][laneId]));
		}
		else if (warpId == 1) {
			u[0] = gU[0][ev[3]]; u[1] = gU[1][ev[3]]; u[2] = gU[2][ev[3]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[2][laneId]));
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[2][laneId]));
			KeU[2] += u[0] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[2][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[0][laneId] + 2.f * RHO[2][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[2][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[2][laneId]));
			KeU[0] += u[2] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[2][laneId]));
			u[0] = gU[0][ev[4]]; u[1] = gU[1][ev[4]]; u[2] = gU[2][ev[4]];
			KeU[2] += u[0] * (LM[1] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[3][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[1] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[1] * (-6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[2] * (-8.f * RHO[0][laneId] + -8.f * RHO[1][laneId] + -8.f * RHO[2][laneId] + -8.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[1] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			u[0] = gU[0][ev[5]]; u[1] = gU[1][ev[5]]; u[2] = gU[2][ev[5]];
			KeU[2] += u[0] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[3][laneId]));
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[1][laneId] + 2.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[3][laneId]));
		}
		else if (warpId == 2) {
			u[0] = gU[0][ev[6]]; u[1] = gU[1][ev[6]]; u[2] = gU[2][ev[6]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[2][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[2][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[2][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[2][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[2][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[2][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[2][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[2][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[2][laneId]));
			u[0] = gU[0][ev[7]]; u[1] = gU[1][ev[7]]; u[2] = gU[2][ev[7]];
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId]));
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[2][laneId] + 2.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			u[0] = gU[0][ev[8]]; u[1] = gU[1][ev[8]]; u[2] = gU[2][ev[8]];
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[3][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[3][laneId]));
		}
		else if (warpId == 3) {
			u[0] = gU[0][ev[9]]; u[1] = gU[1][ev[9]]; u[2] = gU[2][ev[9]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[4][laneId]));
			KeU[1] += u[0] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[4][laneId]));
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[4][laneId]));
			KeU[0] += u[1] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[4][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[4][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[4][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[0][laneId] + 2.f * RHO[4][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[4][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[4][laneId]));
			u[0] = gU[0][ev[10]]; u[1] = gU[1][ev[10]]; u[2] = gU[2][ev[10]];
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[5][laneId]));
			KeU[1] += u[0] * (LM[1] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[1] * (-6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[1] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[2] * (-8.f * RHO[0][laneId] + -8.f * RHO[1][laneId] + -8.f * RHO[4][laneId] + -8.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[1] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId]));
			u[0] = gU[0][ev[11]]; u[1] = gU[1][ev[11]]; u[2] = gU[2][ev[11]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[5][laneId]));
			KeU[1] += u[0] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[1][laneId] + 2.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[5][laneId]));
		}
		else if (warpId == 4) {
			u[0] = gU[0][ev[12]]; u[1] = gU[1][ev[12]]; u[2] = gU[2][ev[12]];
			KeU[2] += u[0] * (LM[1] * (-6.f * RHO[0][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
			KeU[1] += u[0] * (LM[1] * (-6.f * RHO[0][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
			KeU[0] += u[0] * (LM[2] * (-8.f * RHO[0][laneId] + -8.f * RHO[2][laneId] + -8.f * RHO[4][laneId] + -8.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[1] * (6.f * RHO[0][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[1] * (6.f * RHO[0][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
#if 0
			u[0] = gU[0][ev[13]]; u[1] = gU[1][ev[13]]; u[2] = gU[2][ev[13]];
			KeU[2] += u[0] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
#endif
			u[0] = gU[0][ev[14]]; u[1] = gU[1][ev[14]]; u[2] = gU[2][ev[14]];
			KeU[2] += u[0] * (LM[1] * (6.f * RHO[1][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[2] * (-8.f * RHO[1][laneId] + -8.f * RHO[3][laneId] + -8.f * RHO[5][laneId] + -8.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[1] * (6.f * RHO[1][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (-6.f * RHO[1][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[1][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (-6.f * RHO[1][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[5][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[1][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[7][laneId]));

			u[0] = gU[0][ev[15]]; u[1] = gU[1][ev[15]]; u[2] = gU[2][ev[15]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[6][laneId]));
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[6][laneId]));
			KeU[1] += u[0] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[2][laneId] + 2.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[6][laneId]));
		}
		else if (warpId == 5) {
			u[0] = gU[0][ev[16]]; u[1] = gU[1][ev[16]]; u[2] = gU[2][ev[16]];
			KeU[1] += u[0] * (LM[1] * (-6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[2][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[2] * (-8.f * RHO[2][laneId] + -8.f * RHO[3][laneId] + -8.f * RHO[6][laneId] + -8.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (-6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[2][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			u[0] = gU[0][ev[17]]; u[1] = gU[1][ev[17]]; u[2] = gU[2][ev[17]];
			KeU[1] += u[0] * (LM[2] * (-6.f * RHO[3][laneId] + -6.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[3][laneId] + -3.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[3][laneId] + -2.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[3][laneId] + -3.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (-6.f * RHO[3][laneId] + -6.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[3][laneId] + -2.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[3][laneId] + 3.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[3][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[3][laneId] + 2.f * RHO[7][laneId]));
			u[0] = gU[0][ev[18]]; u[1] = gU[1][ev[18]]; u[2] = gU[2][ev[18]];
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[4][laneId]));
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[4][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[4][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[4][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[4][laneId]));
		}
		else if (warpId == 6) {
			u[0] = gU[0][ev[19]]; u[1] = gU[1][ev[19]]; u[2] = gU[2][ev[19]];
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId]));
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[4][laneId] + 2.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[5][laneId]));
			u[0] = gU[0][ev[20]]; u[1] = gU[1][ev[20]]; u[2] = gU[2][ev[20]];
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[5][laneId]));
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[5][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[5][laneId]));
			u[0] = gU[0][ev[21]]; u[1] = gU[1][ev[21]]; u[2] = gU[2][ev[21]];
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[6][laneId]));
			KeU[2] += u[0] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[4][laneId] + 2.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
		}
		else if (warpId == 7) {
			u[0] = gU[0][ev[22]]; u[1] = gU[1][ev[22]]; u[2] = gU[2][ev[22]];
			KeU[2] += u[0] * (LM[1] * (-6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[4][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[4][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (-6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[2] * (-8.f * RHO[4][laneId] + -8.f * RHO[5][laneId] + -8.f * RHO[6][laneId] + -8.f * RHO[7][laneId]));
			u[0] = gU[0][ev[23]]; u[1] = gU[1][ev[23]]; u[2] = gU[2][ev[23]];
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[5][laneId] + -3.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[2] * (-6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[5][laneId] + -2.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[5][laneId] + 2.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[5][laneId] + -3.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (-6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[5][laneId] + -2.f * RHO[7][laneId]));
			u[0] = gU[0][ev[24]]; u[1] = gU[1][ev[24]]; u[2] = gU[2][ev[24]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[6][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[6][laneId]));
			u[0] = gU[0][ev[25]]; u[1] = gU[1][ev[25]]; u[2] = gU[2][ev[25]];
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[6][laneId] + 2.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (-6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[6][laneId] + -3.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[6][laneId] + -2.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (-6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[6][laneId] + -3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[6][laneId] + -2.f * RHO[7][laneId]));
			u[0] = gU[0][ev[26]]; u[1] = gU[1][ev[26]]; u[2] = gU[2][ev[26]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[7][laneId]));
		}
	}

#if 1

	if (warpId >= 4) {
		for (int i = 0; i < 3; i++) sumKeU[i][warpId - 4][laneId] = KeU[i];
	}
	__syncthreads();
	if (warpId < 4) {
		for (int i = 0; i < 3; i++) sumKeU[i][warpId][laneId] += KeU[i];
	}
	__syncthreads();
	if (warpId < 2) {
		for (int i = 0; i < 3; i++) sumKeU[i][warpId][laneId] += sumKeU[i][warpId + 2][laneId];
	}
	__syncthreads();
	if (warpId < 1 && !fiction) {
		for (int i = 0; i < 3; i++) KeU[i] = sumKeU[i][warpId][laneId] + sumKeU[i][warpId + 1][laneId];

		//if (ev[13] == 394689) {
		//	printf("ku = (%.4le, %.4le, %.4le)\n", KeU[0], KeU[1], KeU[2]);
		//}
		//double f[3] = { gF[0][ev[13]], gF[1][ev[13]], gF[2][ev[13]] };
		u[0] = gU[0][ev[13]]; u[1] = gU[1][ev[13]]; u[2] = gU[2][ev[13]];
		//KeU[0] =
		//	u[0] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId])) 
		//	+ u[1] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]))
		//	+ u[2] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
		//KeU[1] += u[0] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]))
		//	+ u[2] * (LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]))
		//	+ u[1] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
		//KeU[2] += u[0] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]))
		//	+ u[1] * (LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]))
		//	+ u[2] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
		float d = LM[3] * 8.f * (RHO[0][laneId] + RHO[1][laneId] + RHO[2][laneId] + RHO[3][laneId] + RHO[4][laneId] + RHO[5][laneId] + RHO[6][laneId] + RHO[7][laneId]);
		float t1 = LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]);
		float t2 = LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]);
		float t3 = LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]);
		u[0] = w * (gF[0][ev[13]] - KeU[0] - u[1] * t1 - u[2] * t2) / d + (1. - w) * u[0];
		u[1] = w * (gF[1][ev[13]] - KeU[1] - u[0] * t1 - u[2] * t3) / d + (1. - w) * u[1];
		u[2] = w * (gF[2][ev[13]] - KeU[2] - u[0] * t2 - u[1] * t3) / d + (1. - w) * u[2];

		if (vflag.is_dirichlet_boundary()) {
			u[0] = 0; u[1] = 0; u[2] = 0;
		}

		gU[0][ev[13]] = u[0]; gU[1][ev[13]] = u[1]; gU[2][ev[13]] = u[2];
	}
#endif
}

void homo::Grid::gs_relaxation_ex(float w_SOR /*= 1.f*/)
{
	if (!is_root) return;
	// change to 8 bytes bank
	use4Bytesbank();
	useGrid_g();
	devArray_t<int, 3>  gridCellReso{};
	devArray_t<int, 8>  gsCellEnd{};
	devArray_t<int, 8>  gsVertexEnd{};
	for (int i = 0; i < 8; i++) {
		gsCellEnd[i] = gsCellSetEnd[i];
		gsVertexEnd[i] = gsVertexSetEnd[i];
		if (i < 3) gridCellReso[i] = cellReso[i];
	}
	for (int i = 0; i < 8; i++) {
		size_t grid_size, block_size;
		int n_gs = gsVertexEnd[i] - (i == 0 ? 0 : gsVertexEnd[i - 1]);
		make_kernel_param(&grid_size, &block_size, n_gs * 8, 32 * 8);
		gs_relaxation_otf_kernel_opt << <grid_size, block_size >> > (i, rho_g, gridCellReso, vertflag, cellflag, w_SOR);
		hipDeviceSynchronize();
		cuda_error_check;
		enforce_period_boundary(u_g);
	}
	enforce_period_boundary(u_g);
	//pad_vertex_data(u_g);
	hipDeviceSynchronize();
	cuda_error_check;

}

__global__ void interpDensityFrom_kernel(float* rholist, hipTextureObject_t rhoTex) {
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	int reso[3] = { gGridCellReso[0],gGridCellReso[1],gGridCellReso[2] };
	int ne = gGridCellReso[0] * gGridCellReso[1] * gGridCellReso[2];
	if (tid >= ne) return;
	int epos[3] = { tid % reso[0], tid / reso[0] % reso[1], tid / (reso[0] * reso[1]) };

	//lexi2gs(int lexpos[3], int gsreso[3][8], int gsend[8], bool padded = false) 
	int gsid = lexi2gs(epos, gGsCellReso, gGsCellEnd);
	//float3 p{ float(epos[0]) / reso[0], float(epos[1]) / reso[1], float(epos[2]) / reso[2] };
	float p[3] = { float(epos[0]) / reso[0], float(epos[1]) / reso[1], float(epos[2]) / reso[2] };
	float f = tex3D<float>(rhoTex, p[0], p[1], p[2]);
	rholist[gsid] = f;
}

void homo::Grid::interpDensityFrom(const std::string& fname, VoxelIOFormat frmat)
{
	useGrid_g();
	std::vector<float> values;
	int reso[3];
	readDensity(fname, values, reso, frmat);

	// Allocate CUDA array in device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* cuArray;
	hipExtent extent{ reso[0],reso[1],reso[2] };
	//hipMalloc3DArray(hipArray_t * array, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int flags __dv(0));
	CheckErr(hipMalloc3DArray(&cuArray, &channelDesc, extent));
	// Copy to device memory some data located at address h_data
    // in host memory hipMemcpy3DParms copyParams = {0};
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(values.data(), reso[0] * sizeof(float), reso[1], reso[2]);
	copyParams.dstArray = cuArray;
	copyParams.extent = extent;
	copyParams.kind = hipMemcpyHostToDevice;
	CheckErr(hipMemcpy3D(&copyParams));
	//CheckErr(hipMemcpyToArray(cuArray, 0, 0, values.data(), values.size(), hipMemcpyHostToDevice)); // [deprecated]

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;
	// Set texture description
	struct hipTextureDesc rhoTexDesc;
	memset(&rhoTexDesc, 0, sizeof(rhoTexDesc));
	rhoTexDesc.addressMode[0] = hipAddressModeBorder;
	rhoTexDesc.addressMode[1] = hipAddressModeBorder;
	rhoTexDesc.addressMode[2] = hipAddressModeBorder;
	rhoTexDesc.filterMode = hipFilterModeLinear;
	rhoTexDesc.readMode = hipReadModeElementType;
	rhoTexDesc.normalizedCoords = 1;
	// create texture object
	hipTextureObject_t rhoTex = 0;
	CheckErr(hipCreateTextureObject(&rhoTex, &resDesc, &rhoTexDesc, NULL));
	
	size_t grid_size, block_size;
	make_kernel_param(&grid_size, &block_size, n_cells(), 256);
	interpDensityFrom_kernel << <grid_size, block_size >> > (rho_g, rhoTex);
	hipDeviceSynchronize();
	cuda_error_check;

	CheckErr(hipDestroyTextureObject(rhoTex));
	CheckErr(hipFreeArray(cuArray));
	cuda_error_check;

	pad_cell_data(rho_g);
}


//template<int BlockSize = 256>
__global__ void update_residual_otf_kernel_opt(
	int nv, float* rholist,
	devArray_t<int, 3> gridCellReso, 
	VertexFlags* vflags, CellFlags* eflags,
	float diag_strength
) {
	__shared__ float LM[5];
	__shared__ float RHO[8][32];
	__shared__ float sumKeU[3][4][32];
	__shared__ float sumKs[3][3];

	// load lam and mu 
	if (threadIdx.x < 5) {
		LM[threadIdx.x] = gLM[threadIdx.x];
	}

	int warpId = threadIdx.x / 32;
	int laneId = threadIdx.x % 32;

	int vid = blockIdx.x * 32 + laneId;
	
	bool fiction = false;

	fiction = vid >= nv;

	GridVertexIndex indexer(gridCellReso[0], gridCellReso[1], gridCellReso[2]);
	VertexFlags vflag;
	if (!fiction) {
		vflag = vflags[vid];
		fiction = fiction || vflag.is_fiction() || vflag.is_period_padding();
		indexer.locate(vid, vflag.get_gscolor(), gGsVertexEnd);
	}

	// load density field
	for (int i = 0; i < 8; i++) { RHO[i][laneId] = 0; }
	if (!fiction) {
		CellFlags eflag;
#pragma unroll
		for (int i = 0; i < 8; i++) {
			int elementId = indexer.neighElement(i, gGsCellEnd, gGsCellReso).getId();
			if (elementId != -1) {
				eflag = eflags[elementId];
			/*	if (!eflag.is_fiction())*/ RHO[i][laneId] = powf(rholist[elementId], exp_penal[0]);
			} 	
		}
	} 
	__syncthreads();

	int ev[27];
	if (!fiction) {
		for (int i = 0; i < 27; i++) {
			VertexFlags nvflag;
			int vneighId = indexer.neighVertex(i, gGsVertexEnd, gGsVertexReso).getId();
			ev[i] = vneighId;
			if (vneighId != -1) {
				nvflag = vflags[vneighId];
			}
		}
	}

	float KeU[3] = { 0. };
	float u[3];
	if (!fiction) {
		if (warpId == 0) {
			u[0] = gU[0][ev[0]]; u[1] = gU[1][ev[0]]; u[2] = gU[2][ev[0]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[0][laneId]));
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[0][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[0][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[0][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[0][laneId]));
			u[0] = gU[0][ev[1]]; u[1] = gU[1][ev[1]]; u[2] = gU[2][ev[1]];
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[0][laneId] + 2.f * RHO[1][laneId]));
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId]));
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId]));
			KeU[2] += u[1] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[1][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[1][laneId]));
			KeU[1] += u[2] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[1][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[1][laneId]));
			u[0] = gU[0][ev[2]]; u[1] = gU[1][ev[2]]; u[2] = gU[2][ev[2]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[1][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[1][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[1][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[1][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[1][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[1][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[1][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[1][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[1][laneId]));
		}
		else if (warpId == 1) {
			u[0] = gU[0][ev[3]]; u[1] = gU[1][ev[3]]; u[2] = gU[2][ev[3]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[2][laneId]));
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[2][laneId]));
			KeU[2] += u[0] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[2][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[0][laneId] + 2.f * RHO[2][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[2][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[2][laneId]));
			KeU[0] += u[2] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[2][laneId]));
			u[0] = gU[0][ev[4]]; u[1] = gU[1][ev[4]]; u[2] = gU[2][ev[4]];
			KeU[2] += u[0] * (LM[1] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[3][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[1] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[1] * (-6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[2] * (-8.f * RHO[0][laneId] + -8.f * RHO[1][laneId] + -8.f * RHO[2][laneId] + -8.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[1] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			u[0] = gU[0][ev[5]]; u[1] = gU[1][ev[5]]; u[2] = gU[2][ev[5]];
			KeU[2] += u[0] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[3][laneId]));
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[1][laneId] + 2.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[3][laneId]));
		}
		else if (warpId == 2) {
			u[0] = gU[0][ev[6]]; u[1] = gU[1][ev[6]]; u[2] = gU[2][ev[6]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[2][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[2][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[2][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[2][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[2][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[2][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[2][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[2][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[2][laneId]));
			u[0] = gU[0][ev[7]]; u[1] = gU[1][ev[7]]; u[2] = gU[2][ev[7]];
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId]));
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[2][laneId] + 2.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[3][laneId]));
			u[0] = gU[0][ev[8]]; u[1] = gU[1][ev[8]]; u[2] = gU[2][ev[8]];
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[3][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[3][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[3][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[3][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[3][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[3][laneId]));
		}
		else if (warpId == 3) {
			u[0] = gU[0][ev[9]]; u[1] = gU[1][ev[9]]; u[2] = gU[2][ev[9]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[4][laneId]));
			KeU[1] += u[0] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[4][laneId]));
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[4][laneId]));
			KeU[0] += u[1] * (LM[2] * (-6.f * RHO[0][laneId] + -6.f * RHO[4][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[0][laneId] + 3.f * RHO[4][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[0][laneId] + -2.f * RHO[4][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[0][laneId] + 2.f * RHO[4][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[4][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[0][laneId] + -3.f * RHO[4][laneId]));
			u[0] = gU[0][ev[10]]; u[1] = gU[1][ev[10]]; u[2] = gU[2][ev[10]];
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[5][laneId]));
			KeU[1] += u[0] * (LM[1] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[1] * (-6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[1] * (-6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[2] * (-8.f * RHO[0][laneId] + -8.f * RHO[1][laneId] + -8.f * RHO[4][laneId] + -8.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[1][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[1][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[1] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId]));
			u[0] = gU[0][ev[11]]; u[1] = gU[1][ev[11]]; u[2] = gU[2][ev[11]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[5][laneId]));
			KeU[1] += u[0] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[1][laneId] + -2.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[2] * (6.f * RHO[1][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[1][laneId] + 2.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[1][laneId] + -3.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[1][laneId] + 3.f * RHO[5][laneId]));
		}
		else if (warpId == 4) {
			u[0] = gU[0][ev[12]]; u[1] = gU[1][ev[12]]; u[2] = gU[2][ev[12]];
			KeU[2] += u[0] * (LM[1] * (-6.f * RHO[0][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
			KeU[1] += u[0] * (LM[1] * (-6.f * RHO[0][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
			KeU[0] += u[0] * (LM[2] * (-8.f * RHO[0][laneId] + -8.f * RHO[2][laneId] + -8.f * RHO[4][laneId] + -8.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[1] * (6.f * RHO[0][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[1] * (6.f * RHO[0][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[0][laneId] + 4.f * RHO[2][laneId] + 4.f * RHO[4][laneId] + 4.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[0][laneId] + -3.f * RHO[2][laneId] + -3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
#if 1
			u[0] = gU[0][ev[13]]; u[1] = gU[1][ev[13]]; u[2] = gU[2][ev[13]];
			KeU[2] += u[0] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + 6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (6.f * RHO[0][laneId] + 6.f * RHO[1][laneId] + -6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (6.f * RHO[0][laneId] + -6.f * RHO[1][laneId] + 6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + -6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[3] * (8.f * RHO[0][laneId] + 8.f * RHO[1][laneId] + 8.f * RHO[2][laneId] + 8.f * RHO[3][laneId] + 8.f * RHO[4][laneId] + 8.f * RHO[5][laneId] + 8.f * RHO[6][laneId] + 8.f * RHO[7][laneId]));
#endif
			u[0] = gU[0][ev[14]]; u[1] = gU[1][ev[14]]; u[2] = gU[2][ev[14]];
			KeU[2] += u[0] * (LM[1] * (6.f * RHO[1][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[2] * (-8.f * RHO[1][laneId] + -8.f * RHO[3][laneId] + -8.f * RHO[5][laneId] + -8.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[1] * (6.f * RHO[1][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (-6.f * RHO[1][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[1][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (-6.f * RHO[1][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[5][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[1][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[1][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[7][laneId]));
		}
		else if (warpId == 5) {
			u[0] = gU[0][ev[15]]; u[1] = gU[1][ev[15]]; u[2] = gU[2][ev[15]];
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[6][laneId]));
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[6][laneId]));
			KeU[1] += u[0] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[2] * (6.f * RHO[2][laneId] + 6.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[2][laneId] + -2.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[2][laneId] + 2.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[2][laneId] + 3.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[2][laneId] + -3.f * RHO[6][laneId]));

			u[0] = gU[0][ev[16]]; u[1] = gU[1][ev[16]]; u[2] = gU[2][ev[16]];
			KeU[1] += u[0] * (LM[1] * (-6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[2][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (6.f * RHO[2][laneId] + 6.f * RHO[3][laneId] + -6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[2] * (-8.f * RHO[2][laneId] + -8.f * RHO[3][laneId] + -8.f * RHO[6][laneId] + -8.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (-6.f * RHO[2][laneId] + -6.f * RHO[3][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[2] * (4.f * RHO[2][laneId] + 4.f * RHO[3][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[2][laneId] + -3.f * RHO[3][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));

			u[0] = gU[0][ev[17]]; u[1] = gU[1][ev[17]]; u[2] = gU[2][ev[17]];
			KeU[1] += u[0] * (LM[2] * (-6.f * RHO[3][laneId] + -6.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[1] * (3.f * RHO[3][laneId] + -3.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[3][laneId] + -2.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (3.f * RHO[3][laneId] + -3.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (-6.f * RHO[3][laneId] + -6.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[3][laneId] + -2.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (-3.f * RHO[3][laneId] + 3.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (-3.f * RHO[3][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[0] * (2.f * RHO[3][laneId] + 2.f * RHO[7][laneId]));
			u[0] = gU[0][ev[18]]; u[1] = gU[1][ev[18]]; u[2] = gU[2][ev[18]];
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[4][laneId]));
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[4][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[4][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[4][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[4][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[4][laneId]));
		}
		else if (warpId == 6) {
			u[0] = gU[0][ev[19]]; u[1] = gU[1][ev[19]]; u[2] = gU[2][ev[19]];
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId]));
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[4][laneId] + 2.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[5][laneId]));

			u[0] = gU[0][ev[20]]; u[1] = gU[1][ev[20]]; u[2] = gU[2][ev[20]];
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[5][laneId]));
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[5][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[5][laneId]));
			KeU[2] += u[1] * (LM[2] * (3.f * RHO[5][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[5][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[5][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[5][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[5][laneId]));
			KeU[1] += u[2] * (LM[2] * (3.f * RHO[5][laneId]));

			u[0] = gU[0][ev[21]]; u[1] = gU[1][ev[21]]; u[2] = gU[2][ev[21]];
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[6][laneId]));
			KeU[2] += u[0] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[4][laneId] + 3.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[4][laneId] + 2.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[4][laneId] + -2.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[4][laneId] + -3.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[2] * (6.f * RHO[4][laneId] + 6.f * RHO[6][laneId]));

			u[0] = gU[0][ev[22]]; u[1] = gU[1][ev[22]]; u[2] = gU[2][ev[22]];
			KeU[2] += u[0] * (LM[1] * (-6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[2] * (4.f * RHO[4][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[2] * (4.f * RHO[4][laneId] + 4.f * RHO[5][laneId] + 4.f * RHO[6][laneId] + 4.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (-6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + 6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[4][laneId] + -3.f * RHO[5][laneId] + -3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (6.f * RHO[4][laneId] + -6.f * RHO[5][laneId] + 6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (6.f * RHO[4][laneId] + 6.f * RHO[5][laneId] + -6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[2] * (-8.f * RHO[4][laneId] + -8.f * RHO[5][laneId] + -8.f * RHO[6][laneId] + -8.f * RHO[7][laneId]));
		}
		else if (warpId == 7) {
			u[0] = gU[0][ev[23]]; u[1] = gU[1][ev[23]]; u[2] = gU[2][ev[23]];
			KeU[1] += u[0] * (LM[1] * (3.f * RHO[5][laneId] + -3.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[2] * (-6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[0] * (LM[4] * (-2.f * RHO[5][laneId] + -2.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[0] * (2.f * RHO[5][laneId] + 2.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[1] * (-3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (-3.f * RHO[5][laneId] + 3.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[1] * (3.f * RHO[5][laneId] + -3.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (-6.f * RHO[5][laneId] + -6.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[5][laneId] + -2.f * RHO[7][laneId]));
			u[0] = gU[0][ev[24]]; u[1] = gU[1][ev[24]]; u[2] = gU[2][ev[24]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[6][laneId]));
			KeU[2] += u[0] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[1] += u[0] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[6][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[6][laneId]));
			KeU[0] += u[1] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[0] += u[2] * (LM[2] * (3.f * RHO[6][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[6][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[6][laneId]));
			u[0] = gU[0][ev[25]]; u[1] = gU[1][ev[25]]; u[2] = gU[2][ev[25]];
			KeU[0] += u[0] * (LM[0] * (2.f * RHO[6][laneId] + 2.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[1] * (-3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[1] * (-3.f * RHO[6][laneId] + 3.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (-6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[1] * (3.f * RHO[6][laneId] + -3.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[4] * (-2.f * RHO[6][laneId] + -2.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (-6.f * RHO[6][laneId] + -6.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[1] * (3.f * RHO[6][laneId] + -3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[4] * (-2.f * RHO[6][laneId] + -2.f * RHO[7][laneId]));
			u[0] = gU[0][ev[26]]; u[1] = gU[1][ev[26]]; u[2] = gU[2][ev[26]];
			KeU[0] += u[0] * (LM[3] * (-2.f * RHO[7][laneId]));
			KeU[1] += u[0] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[2] += u[0] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[1] += u[1] * (LM[3] * (-2.f * RHO[7][laneId]));
			KeU[2] += u[1] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[0] += u[1] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[1] += u[2] * (LM[2] * (-3.f * RHO[7][laneId]));
			KeU[2] += u[2] * (LM[3] * (-2.f * RHO[7][laneId]));
			KeU[0] += u[2] * (LM[2] * (-3.f * RHO[7][laneId]));
		}
	}

#if 1

	if (warpId >= 4) {
		for (int i = 0; i < 3; i++) sumKeU[i][warpId - 4][laneId] = KeU[i];
	}
	__syncthreads();
	if (warpId < 4) {
		for (int i = 0; i < 3; i++) sumKeU[i][warpId][laneId] += KeU[i];
	}
	__syncthreads();
	if (warpId < 2) {
		for (int i = 0; i < 3; i++) sumKeU[i][warpId][laneId] += sumKeU[i][warpId + 2][laneId];
	}
	__syncthreads();
	if (warpId < 1 && !fiction) {
		for (int i = 0; i < 3; i++) KeU[i] = sumKeU[i][warpId][laneId] + sumKeU[i][warpId + 1][laneId];

		float r[3];
		r[0] = gF[0][ev[13]] - KeU[0]; r[1] = gF[1][ev[13]] - KeU[1]; r[2] = gF[2][ev[13]] - KeU[2];

		if (vflag.is_dirichlet_boundary()) { r[0] = 0; r[1] = 0; r[2] = 0; }

		gR[0][ev[13]] = r[0]; gR[1][ev[13]] = r[1]; gR[2][ev[13]] = r[2];
	}
#endif
}

void homo::Grid::update_residual_ex(void)
{
	useGrid_g();
	devArray_t<int, 3> gridCellReso{ cellReso[0],cellReso[1],cellReso[2] };
	VertexFlags* vflags = vertflag;
	CellFlags* eflags = cellflag;
	if (assemb_otf) {
		size_t grid_size, block_size;
		make_kernel_param(&grid_size, &block_size, n_gsvertices() * 8, 32 * 8);
		update_residual_otf_kernel_opt << <grid_size, block_size >> > (n_gsvertices(), rho_g, gridCellReso,
			vflags, eflags, diag_strength);
		hipDeviceSynchronize();
		cuda_error_check;
	}
	pad_vertex_data(r_g);
}

// gather per fine element matrix to coarse stencil, one thread for one coarse vertex 
// stencil was organized in lexico order(No padding), and should be transferred to gs order
//template<int BlockSize = 256>
__global__ void restrict_stencil_otf_aos_kernel_1(
	int nv, float* rholist, CellFlags* eflags, VertexFlags* vflags
) {
	//__shared__ glm::mat<3, 3, double> KE[8][8];
	__shared__ glm::mat3 KE[8][8];
	__shared__ int coarseReso[3];
	__shared__ int fineReso[3];
	
	if (threadIdx.x < 3) { 
		coarseReso[threadIdx.x] = gGridCellReso[threadIdx.x]; 
		fineReso[threadIdx.x] = coarseReso[threadIdx.x] * gUpCoarse[threadIdx.x];
	}
	loadTemplateMatrix(KE);

	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	int coarseRatio[3] = { gUpCoarse[0], gUpCoarse[1], gUpCoarse[2] };
	int vipos[3] = {
		tid % (coarseReso[0] + 1),
		tid / (coarseReso[0] + 1) % (coarseReso[1] + 1),
		tid / ((coarseReso[0] + 1) * (coarseReso[1] + 1)) };
	//size_t vid = lexi2gs(vipos, gGsVertexReso, gGsVertexEnd);
	size_t vid = tid;

	//bool debug = vid == 63;
	bool debug = false;

	if (vid >= nv) return;

	vipos[0] *= coarseRatio[0]; vipos[1] *= coarseRatio[1]; vipos[2] *= coarseRatio[2];

	float pr = coarseRatio[0] * coarseRatio[1] * coarseRatio[2];

	if (debug) { printf("vipos = (%d, %d, %d)\n", vipos[0], vipos[1], vipos[2]); }

	for (int vj = 0; vj < 27; vj++) {
		int coarse_vj_off[3] = {
			coarseRatio[0] * (vj % 3 -1),
			coarseRatio[1] * (vj / 3 % 3 -1),
			coarseRatio[2] * (vj / 9 - 1)
		};
		//glm::mat<3, 3, double> st(0.f);
		glm::mat3 st(0.f);
		if (debug) { printf("coarse_vj_off = (%d, %d, %d)\n", coarse_vj_off[0], coarse_vj_off[1], coarse_vj_off[2]); }
		for (int xfine_off = -coarseRatio[0]; xfine_off < coarseRatio[0]; xfine_off++) {
			for (int yfine_off = -coarseRatio[1]; yfine_off < coarseRatio[1]; yfine_off++) {
				for (int zfine_off = -coarseRatio[2]; zfine_off < coarseRatio[2]; zfine_off++) {
					int e_fine_off[3] = {
						coarse_vj_off[0] + xfine_off,
						coarse_vj_off[1] + yfine_off,
						coarse_vj_off[2] + zfine_off,
					};
					// exclude elements out of neighborhood
					if (e_fine_off[0] < -coarseRatio[0] || e_fine_off[0] >= coarseRatio[0] ||
						e_fine_off[1] < -coarseRatio[1] || e_fine_off[1] >= coarseRatio[1] ||
						e_fine_off[2] < -coarseRatio[2] || e_fine_off[2] >= coarseRatio[2]) {
						continue;
					};
					if (debug) { printf(" e_fine_off = (%d, %d, %d)\n", e_fine_off[0], e_fine_off[1], e_fine_off[2]); }
					int e_fine_pos[3] = {
						vipos[0] + e_fine_off[0], vipos[1] + e_fine_off[1], vipos[2] + e_fine_off[2]
					};
					// exclude padded element
					if (e_fine_pos[0] < 0 || e_fine_pos[0] >= fineReso[0] ||
						e_fine_pos[1] < 0 || e_fine_pos[1] >= fineReso[1] ||
						e_fine_pos[2] < 0 || e_fine_pos[2] >= fineReso[2]) {
						continue;
					}
					int eid = lexi2gs(e_fine_pos, gGsFineCellReso, gGsFineCellEnd);
					//auto eflag = eflags[eid];
					float rho_penal = powf(rholist[eid], exp_penal[0]);
					if (debug) { printf(" e_fine_pos = (%d, %d, %d), eid = %d, rhopenal = %f\n", e_fine_pos[0], e_fine_pos[1], e_fine_pos[2], eid, rho_penal); }
					for (int e_vi = 0; e_vi < 8; e_vi++) {
						int e_vi_fine_off[3] = {
							e_fine_off[0] + e_vi % 2,
							e_fine_off[1] + e_vi / 2 % 2,
							e_fine_off[2] + e_vi / 4
						};
						if (!inStrictBound(e_vi_fine_off, coarseRatio)) continue;
						float wi = (coarseRatio[0] - abs(e_vi_fine_off[0])) *
							(coarseRatio[1] - abs(e_vi_fine_off[1])) *
							(coarseRatio[2] - abs(e_vi_fine_off[2])) / pr;
						if (debug) printf("   e_vi_off = (%d, %d, %d), wi = %f\n", e_vi_fine_off[0], e_vi_fine_off[1], e_vi_fine_off[2], wi);
						wi *= rho_penal;
						for (int e_vj = 0; e_vj < 8; e_vj++) {
							int vij_off[3] = {
								abs(e_fine_off[0] + e_vj % 2 - coarse_vj_off[0]),
								abs(e_fine_off[1] + e_vj / 2 % 2 - coarse_vj_off[1]),
								abs(e_fine_off[2] + e_vj / 4 - coarse_vj_off[2])
							};
							if (vij_off[0] >= coarseRatio[0] || vij_off[1] >= coarseRatio[1] ||
								vij_off[2] >= coarseRatio[2]) {
								continue;
							}
							float wj = (coarseRatio[0] - vij_off[0]) *
								(coarseRatio[1] - vij_off[1]) *
								(coarseRatio[2] - vij_off[2]) / pr;
							if (debug) printf("    vij_off = (%d, %d, %d), wi = %f\n", vij_off[0], vij_off[1], vij_off[2], wj);
							st += (wi * wj) * KE[e_vi][e_vj];
						}
					}
				}
			}
		}
		rxstencil[vj][vid] = st;
	}
}

// one thread of one coarse vertex
//template<int BlockSize = 256>
__global__ void restrict_stencil_aos_kernel_1(
	int nv_coarse, int nv_fine,
	VertexFlags* vflags,
	VertexFlags* vfineflags
) {
	__shared__ int gsVertexEnd[8];
	__shared__ int gsFineVertexEnd[8];
	__shared__ int gsFineVertexReso[3][8];

	if (threadIdx.x < 24) {
		gsFineVertexReso[threadIdx.x / 8][threadIdx.x % 8] = gGsFineVertexReso[threadIdx.x / 8][threadIdx.x % 8];
	}
	if (threadIdx.x < 8) {
		gsVertexEnd[threadIdx.x] = gGsVertexEnd[threadIdx.x];
		gsFineVertexEnd[threadIdx.x] = gGsFineVertexEnd[threadIdx.x];
	}
	__syncthreads();

	bool fiction = false;
	int laneId = threadIdx.x % 32;
	int warpId = threadIdx.x / 32;
	//size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	//size_t vid = blockIdx.x * 32 + laneId;
	size_t vid = blockIdx.x * blockDim.x + threadIdx.x;

	if (vid >= nv_coarse) fiction = true;

	VertexFlags vflag;
	if (!fiction) { 
		vflag = vflags[vid]; 
		fiction = vflag.is_fiction();
	}

	int coarseRatio[3] = { gUpCoarse[0], gUpCoarse[1], gUpCoarse[2] };

	float pr = coarseRatio[0] * coarseRatio[1] * coarseRatio[2];

	GridVertexIndex indexer(gGridCellReso[0], gGridCellReso[1], gGridCellReso[2]);
	indexer.locate(vid, vflag.get_gscolor(), gsVertexEnd);

	bool nondyadic = coarseRatio[0] > 2 || coarseRatio[1] > 2 || coarseRatio[2] > 2;

	if (!fiction && !vflag.is_period_padding()) {
		for (int i = 0; i < 27; i++) {
			int coarse_vj_off[3] = {
				coarseRatio[0] * (i % 3 - 1),
				coarseRatio[1] * (i / 3 % 3 - 1),
				coarseRatio[2] * (i / 9 - 1)
			};
			glm::mat3 st(0.f);
			for (int xfine_off = -coarseRatio[0]; xfine_off <= coarseRatio[0]; xfine_off++) {
				for (int yfine_off = -coarseRatio[1]; yfine_off <= coarseRatio[1]; yfine_off++) {
					for (int zfine_off = -coarseRatio[2]; zfine_off <= coarseRatio[2]; zfine_off++) {
						int vi_fine_off[3] = {
							xfine_off + coarse_vj_off[0],
							yfine_off + coarse_vj_off[1],
							zfine_off + coarse_vj_off[2]
						};
						if (!inStrictBound(vi_fine_off, coarseRatio)) continue;
						int vi_neighId;
						if (nondyadic) {
							vi_neighId = indexer.neighFineVertex(vi_fine_off, coarseRatio, gsFineVertexEnd, gsFineVertexReso, true).getId();
						} else {
							vi_neighId = indexer.neighFineVertex(vi_fine_off, coarseRatio, gsFineVertexEnd, gsFineVertexReso, false).getId();
						}
						float wi = (coarseRatio[0] - abs(vi_fine_off[0])) *
							(coarseRatio[1] - abs(vi_fine_off[1])) *
							(coarseRatio[2] - abs(vi_fine_off[2])) / pr;
						for (int vj_offid = 0; vj_offid < 27; vj_offid++) {
							int vij_off[3] = {
								abs(vi_fine_off[0] + vj_offid % 3 - 1 - coarse_vj_off[0]),
								abs(vi_fine_off[1] + vj_offid / 3 % 3 - 1 - coarse_vj_off[1]),
								abs(vi_fine_off[2] + vj_offid / 9 - 1 - coarse_vj_off[2]) 
							};
							if (vij_off[0] >= coarseRatio[0] || vij_off[1] >= coarseRatio[1] || vij_off[2] >= coarseRatio[2]) {
								continue;
							}
							float wj = (coarseRatio[0] - vij_off[0]) * 
								(coarseRatio[1] - vij_off[1]) * 
								(coarseRatio[2] - vij_off[2]) / pr;
							st += wi * wj * rxFineStencil[vj_offid][vi_neighId];
						}
					}
				}
			}
			rxstencil[i][vid] = st;
		}
	}
}
